#include "hip/hip_runtime.h"
//
// Configuration
//

// Header
#include "nos.hpp"

// Standard library
#include <iostream>

// Local
#include "../global.hpp"
#include "../logger.hpp"
#include "../cudahelper/chrono.hpp"


//
// Kernels
//

__device__ float interpolate_kernel(
                const Eigen::Map<const Eigen::MatrixXf> &source,
                const Point<float>::type &p)
{
        // Get fractional and integral part of the coordinates
        const int x_int = (int) p.x();
        const int y_int = (int) p.y();
        const float x_fract = p.x() - x_int;
        const float y_fract = p.y() - y_int;

        return    source(y_int, x_int)     * (1-x_fract)*(1-y_fract)
                + source(y_int, x_int+1)   * x_fract*(1-y_fract)
                + source(y_int+1, x_int)   * (1-x_fract)*y_fract
                + source(y_int+1, x_int+1) * x_fract*y_fract;

}

__constant__ float _transform[4];

__global__ void rotate_kernel(const float *_input, float *_output)
{
        // Compute the thread dimensions
        const int col = blockIdx.x;
        const int cols = gridDim.x;
        const int row = threadIdx.y;
        const int rows = blockDim.y;

        // Construct Eigen objects
        Eigen::Map<const Eigen::MatrixXf> input(_input, rows, cols);
        Eigen::Map<Eigen::MatrixXf> output(_output, rows, cols);
        Eigen::Map<const Eigen::Matrix2f> transform(_transform);

        // Calculate the source location
        Point<float>::type origin(cols / 2.0, rows / 2.0);
        Point<float>::type p(col, row);
        Point<float>::type q = ((p - origin) * transform) + origin;

        // Interpolate the source value
        if (q.x() >= 0 && q.x() < cols - 1 && q.y() >= 0
                        && q.y() < rows - 1)
                output(row, col) = interpolate_kernel(input, q);
        else if (col < cols && row < rows)
                output(row, col) = 0;
}


//
// Wrappers
//

void rotate(const CUDAHelper::GlobalMemory<float> *input,
                CUDAHelper::GlobalMemory<float> *output, float angle)
{
        const int rows = input->size(0);
        const int cols = input->size(1);

        // Set-up
        CUDAHelper::Chrono chrono;
        chrono.start();

        // Calculate transform matrix
        Eigen::Matrix2f transform_data;
        transform_data <<       std::cos(angle), -std::sin(angle),
                                std::sin(angle),  std::cos(angle);
        CUDAHelper::ConstantMemory<float> *transform = new CUDAHelper::ConstantMemory<float>(_transform, CUDAHelper::size_2d(2, 2));
        transform->upload(transform_data.data());

        // Launch
        dim3 threads(1, rows);
        dim3 blocks(cols, 1);
        rotate_kernel<<<blocks, threads>>>(*input, *output);
        CUDAHelper::checkState();

        // Clean-up
        chrono.stop();
        clog(trace) << "Rotation kernel took " << chrono.elapsed() << " ms." << std::endl;
}
