#include "hip/hip_runtime.h"
//
// Configuration
//

// Header
#include "stats.hpp"

// Local
#include "scan.cu"


//
// Kernels
//

__global__ void zscore_kernel(const float *input, float *output) {
    // Shared memory
    extern __shared__ float temp[];

    // Compute the thread dimensions
    const int col = blockIdx.x;
    const int row = threadIdx.y;
    const int rows = blockDim.y;

    // Offsets into shared memory
    float *mean = &temp[2 * rows];
    float *stdev = &temp[2 * rows + 1];

    // Fetch
    temp[row] = input[row + col * rows];
    __syncthreads();

    // Scan to integrate
    scan_array(temp, row, rows, SUM);

    // Calculate the arithmetic mean
    if (row == rows - 1)
        *mean = temp[rows + row] / rows;
    __syncthreads();

    // Fetch and differentiate against mean
    float diff = input[row + col * rows] - *mean;
    temp[row] = diff * diff;

    // Scan to integrate
    scan_array(temp, row, rows, SUM);

    // Calculate the standard deviation
    if (row == rows - 1)
        *stdev = std::sqrt(temp[rows + row] / (rows - 1));
    __syncthreads();

    // Normalize the input
    output[row + col * rows] = (input[row + col * rows] - *mean) / *stdev;
}


//
// Wrappers
//

CUDAHelper::GlobalMemory<float> *
zscore(const CUDAHelper::GlobalMemory<float> *input) {
    // Calculate the z-score
    CUDAHelper::GlobalMemory<float> *output =
        new CUDAHelper::GlobalMemory<float>(input->sizes());
    {
        dim3 threads(1, input->rows());
        dim3 blocks(1, 1);
        zscore_kernel <<<blocks, threads, 2 * input->rows() * sizeof(float) +
                                              2 * sizeof(float)>>>
            (*input, *output);
        CUDAHelper::checkState();
    }

    return output;
}
